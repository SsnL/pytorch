
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathPointwise.cu"
#else

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, NTYPE)           \
  struct Tensor_##NAME##_##NTYPE##_Op {                                 \
    __device__ __forceinline__ void operator()(ntype* out, ntype* in) const { \
      *out = CFUNC(*in);                                                \
    }                                                                   \
                                                                        \
    __device__ __forceinline__ void operator()(ntype* v) const {         \
      *v = CFUNC(*v);                                                   \
    }                                                                   \
  };                                                                    \
                                                                        \
  void THCTensor_(NAME)(THCState* state, THCTensor* self_, THCTensor* src) { \
    THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));               \
    if (self_ == src) {                                                 \
      if (!THC_pointwiseApply1(state, self_, Tensor_##NAME##_##NTYPE##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    } else {                                                            \
      THCTensor_(resizeAs)(state, self_, src);                          \
                                                                        \
      if (!THC_pointwiseApply2(state, self_, src, Tensor_##NAME##_##NTYPE##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    }                                                                   \
                                                                        \
    THCudaCheck(cudaGetLastError());                                    \
  }

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(NAME, CFUNC, NTYPE) \
  IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, NTYPE)

#if defined(THC_NTYPE_IS_FLOAT) || defined(THC_NTYPE_IS_DOUBLE) || defined(THC_NTYPE_IS_HALF)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  log, THCNumerics<ntype>::log,   Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(lgamma, THCNumerics<ntype>::lgamma, Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log1p, THCNumerics<ntype>::log1p, Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  exp, THCNumerics<ntype>::exp,   Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(expm1, THCNumerics<ntype>::expm1, Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cos, THCNumerics<ntype>::cos,   Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  sin, THCNumerics<ntype>::sin,   Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( sqrt, THCNumerics<ntype>::sqrt,  Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(rsqrt, THCNumerics<ntype>::rsqrt, Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( ceil, THCNumerics<ntype>::ceil,  Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(floor, THCNumerics<ntype>::floor, Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(trunc, THCNumerics<ntype>::trunc, Ntype)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  acos, THCNumerics<ntype>::acos,  Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cosh, THCNumerics<ntype>::cosh,  Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  asin, THCNumerics<ntype>::asin,  Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  sinh, THCNumerics<ntype>::sinh,  Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(   tan, THCNumerics<ntype>::tan,   Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  atan, THCNumerics<ntype>::atan,  Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  tanh, THCNumerics<ntype>::tanh,  Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(   erf, THCNumerics<ntype>::erf,   Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(erfinv, THCNumerics<ntype>::erfinv,Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( round, THCNumerics<ntype>::round, Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  frac, THCNumerics<ntype>::frac,  Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cinv, THCNumerics<ntype>::cinv,  Ntype)

#endif

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  neg, THCNumerics<ntype>::neg,   Ntype)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  abs, THCNumerics<ntype>::abs,   Ntype)

#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_
#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC

void THCTensor_(sign)(THCState* state, THCTensor* self_, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorSignOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorSignOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(clamp)(THCState *state, THCTensor *self_, THCTensor *src, ntype min_value,
  ntype max_value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorClampOp<ntype>(min_value, max_value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorClampOp<ntype>(min_value, max_value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cross)(THCState *state, THCTensor *self, THCTensor *x, THCTensor *y, int dimension)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, x, y));

  int i;
  int nd = THCTensor_(nDimension)(state, x);
  ptrdiff_t nelem = THCTensor_(nElement)(state, x);
  THArgCheck(nd == THCTensor_(nDimension)(state, y), 1, "tensors must have same number of dimensions");
  for (i = 0; i < nd; i++) {
    THArgCheck(THCTensor_(size)(state, x, i) == THCTensor_(size)(state, y, i), 1, "dimension %i of x and y does not match", i);
    if (dimension < 0 && THCTensor_(size)(state, x, i) == 3) {
      dimension = i;
    }
  }

  THArgCheck(dimension >= 0 && dimension < nd, 3, "dimension %d out of range", dimension+1);
  THArgCheck(THCTensor_(size)(state, x, dimension) == 3, 3,
      "dimension %d does not have size 3", dimension+1);
  THCTensor_(resizeAs)(state, self, x);

  int64_t sx = THCTensor_(stride)(state, x, dimension);
  int64_t sy = THCTensor_(stride)(state, y, dimension);
  int64_t so = THCTensor_(stride)(state, self, dimension);
  THCTensor *nx = THCTensor_(newNarrow)(state, x, dimension, 0, 1);
  THCTensor *ny = THCTensor_(newNarrow)(state, y, dimension, 0, 1);
  THCTensor *nself = THCTensor_(newNarrow)(state, self, dimension, 0, 1);
  if (!THC_pointwiseApply3(state, nself, nx, ny, TensorCrossOp<ntype>(sx, sy, so))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }
  THCTensor_(free)(state, nx);
  THCTensor_(free)(state, ny);
  THCTensor_(free)(state, nself);
}

#if defined(THC_NTYPE_IS_FLOAT) || defined(THC_NTYPE_IS_DOUBLE) || defined(THC_NTYPE_IS_HALF)

void THCTensor_(atan2)(THCState *state, THCTensor *self_, THCTensor *tx, THCTensor *ty)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, tx, ty));
  THArgCheck(THCTensor_(nElement)(state, tx) ==
             THCTensor_(nElement)(state, ty), 3, "sizes do not match");
  THCTensor_(resizeAs)(state, self_, tx);

  if (!THC_pointwiseApply3(state, self_, tx, ty, TensorATan2Op<ntype>())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(sigmoid)(THCState* state, THCTensor* self_, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorSigmoidOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorSigmoidOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(digamma)(THCState* state, THCTensor* self_, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ != src) {
    THCTensor_(resizeAs)(state, self_, src);
  }
  if (!THC_pointwiseApply2(state, self_, src, TensorDigammaOp<ntype, accntype>())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(polygamma)(THCState* state, THCTensor* self_, int64_t n, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ != src) {
    THCTensor_(resizeAs)(state, self_, src);
  }
  switch (n) {
    case 0:
      if (!THC_pointwiseApply2(state, self_, src, TensorDigammaOp<ntype, accntype>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
      break;
    case 1:
      if (!THC_pointwiseApply2(state, self_, src, TensorTrigammaOp<ntype, accntype>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
      break;
    default:
      THError("polygamma(n,x) is not implemented for n>=2");
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(pow)(THCState *state, THCTensor *self_, THCTensor *src, ntype value) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (THCNumerics<ntype>::eq(value, ScalarConvert<int, ntype>::to(1))) {
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<ntype, 1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<ntype>::eq(value, ScalarConvert<int, ntype>::to(2))) {
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<ntype, 2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<ntype>::eq(value, ScalarConvert<int, ntype>::to(3))) {
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<ntype, 3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<ntype>::eq(value, ScalarConvert<int, ntype>::to(-1))) {
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<ntype, -1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<ntype>::eq(value, ScalarConvert<int, ntype>::to(-2))) {
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<ntype, -2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // fallback implementation using pow
      if (!THC_pointwiseApply1(state, self_, TensorPowOp<ntype, -3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (THCNumerics<ntype>::eq(value, ScalarConvert<int, ntype>::to(1))) {
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<ntype, 1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<ntype>::eq(value, ScalarConvert<int, ntype>::to(2))) {
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<ntype, 2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<ntype>::eq(value, ScalarConvert<int, ntype>::to(3))) {
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<ntype, 3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<ntype>::eq(value, ScalarConvert<int, ntype>::to(-1))) {
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<ntype, -1>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else if (THCNumerics<ntype>::eq(value, ScalarConvert<int, ntype>::to(-2))) {
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<ntype, -2>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // fallback implementation using pow
      if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<ntype, -3>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(tpow)(THCState *state, THCTensor *self_, ntype value, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorTPowOp<ntype>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorTPowOp<ntype>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(lerp)(THCState *state, THCTensor *result, THCTensor *a, THCTensor *b, ntype w)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, result, a, b));
  THArgCheck(THCTensor_(nElement)(state, a) ==
             THCTensor_(nElement)(state, b), 3, "sizes do not match");
  THCTensor_(resizeAs)(state, result, a);

  if (!THC_pointwiseApply3(state, result, a, b, TensorLerpOp<ntype>(w))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

#endif

THC_API void
THCTensor_(cadd)(THCState *state, THCTensor *self_, THCTensor* src1, ntype value, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    if (value == ScalarConvert<int, ntype>::to(1)) {
      // self += src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorAddOp<ntype>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self += value * src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorCAddOp<ntype>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    if (value == ScalarConvert<int, ntype>::to(1)) {
      // self = src1 + src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddOp<ntype>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self = src1 + value * src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorCAddOp<ntype>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(csub)(THCState *state, THCTensor *self_, THCTensor* src1, ntype value, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    if (value == ScalarConvert<int, ntype>::to(1)) {
      // self -= src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorSubOp<ntype>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self += -value * src2
      if (!THC_pointwiseApply2(state, self_, src2,
                                   TensorCAddOp<ntype>(
                                     ScalarNegate<ntype>::to(value)))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    if (value == ScalarConvert<int, ntype>::to(1)) {
      // self = src1 - src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorSubOp<ntype>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self = src1 - value * src2
      if (!THC_pointwiseApply3(state, self_, src1, src2,
                                   TensorCAddOp<ntype>(
                                     ScalarNegate<ntype>::to(value)))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cmul)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self *= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorMulOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 * src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorMulOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cpow)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self = pow(self, src2)
    if (!THC_pointwiseApply2(state, self_, src2, TensorCPowOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = pow(src1, src2)
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorCPowOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cdiv)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorDivOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorDivOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(clshift)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_NTYPE_IS_HALF)
  return THError("clshift not supported for torch.CudaHalfTensor");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorLShiftOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorLShiftOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

THC_API void
THCTensor_(crshift)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_NTYPE_IS_HALF)
  return THError("crshift not supported for torch.CudaHalfTensor");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorRShiftOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorRShiftOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

THC_API void
THCTensor_(cmax)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorMaxOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorMaxOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cmin)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorMinOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorMinOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cremainder)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorCRemainderOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorCRemainderOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cfmod)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorCFmodOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorCFmodOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cmaxValue)(THCState *state, THCTensor *self, THCTensor *src, ntype value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1(state, self, TensorMaxValueOp<ntype>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2(state, self, src, TensorMaxValueOp<ntype>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cminValue)(THCState *state, THCTensor *self, THCTensor *src, ntype value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1(state, self, TensorMinValueOp<ntype>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2(state, self, src, TensorMinValueOp<ntype>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(addcmul)(THCState *state, THCTensor *self_, THCTensor *t, ntype value, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 4, self_, t, src1, src2));
  if(self_ != t)
  {
    THCTensor_(resizeAs)(state, self_, t);
    THCTensor_(copy)(state, self_, t);
  }
  else
  {
    THArgCheck(THCTensor_(nElement)(state, self_) == THCTensor_(nElement)(state, src1),
               1, "sizes do not match");
  }

  THArgCheck(THCTensor_(nElement)(state, src1) == THCTensor_(nElement)(state, src2),
             3, "sizes do not match");

  if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddCMulOp<ntype>(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(addcdiv)(THCState *state, THCTensor *self_, THCTensor *t, ntype value, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 4, self_, t, src1, src2));
  if(self_ != t)
  {
    THCTensor_(resizeAs)(state, self_, t);
    THCTensor_(copy)(state, self_, t);
  }
  else
  {
    THArgCheck(THCTensor_(nElement)(state, self_) == THCTensor_(nElement)(state, src1),
               1, "sizes do not match");
  }
  THArgCheck(THCTensor_(nElement)(state, src1) == THCTensor_(nElement)(state, src2),
             3, "sizes do not match");

  if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddCDivOp<ntype>(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cbitand)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_NTYPE_IS_HALF) || defined(THC_NTYPE_IS_FLOAT) || defined(THC_NTYPE_IS_DOUBLE)
  return THError("cbitand is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorBitAndOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorBitAndOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

THC_API void
THCTensor_(cbitor)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_NTYPE_IS_HALF) || defined(THC_NTYPE_IS_FLOAT) || defined(THC_NTYPE_IS_DOUBLE)
  return THError("cbitor is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorBitOrOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorBitOrOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

THC_API void
THCTensor_(cbitxor)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_NTYPE_IS_HALF) || defined(THC_NTYPE_IS_FLOAT) || defined(THC_NTYPE_IS_DOUBLE)
  return THError("cbitor is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorBitXorOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorBitXorOp<ntype>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}
#endif
