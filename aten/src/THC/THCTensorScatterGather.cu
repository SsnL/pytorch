#include "hip/hip_runtime.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCAtomics.cuh"
#include "THCApply.cuh"

// Compute the offsets into the given tensors for a linear index. For the 't2'
// tensor, dimension 'dim' is skipped. The tensors are assumed to have the same
// size (with the exception of 't2' in dimension 'dim').
// This version uses a static number of dimensions.
template <typename IndexType, typename Ntype, int Dims>
struct IndexToScatterGatherOffsets {
  static __device__ void compute(
      IndexType linearId, const int dim,
      const TensorInfo<int64_t, IndexType>& index, IndexType* indexOffset,
      const TensorInfo<Ntype, IndexType>& t1, IndexType* t1Offset,
      const TensorInfo<Ntype, IndexType>& t2, IndexType* t2Offset) {
    for (int d = Dims - 1; d >= 0; d--) {
      IndexType curDimIndex = linearId % index.sizes[d];
      *indexOffset += curDimIndex * index.strides[d];
      *t1Offset += curDimIndex * t1.strides[d];
      if (d != dim) {
        *t2Offset += curDimIndex * t2.strides[d];
      }
      linearId /= index.sizes[d];
    }
  }

  static __device__ void compute(
      IndexType linearId, const int dim,
      const TensorInfo<int64_t, IndexType>& index, IndexType* indexOffset,
      const TensorInfo<Ntype, IndexType>& t2, IndexType* t2Offset) {
    for (int d = Dims - 1; d >= 0; d--) {
      IndexType curDimIndex = linearId % index.sizes[d];
      *indexOffset += curDimIndex * index.strides[d];
      if (d != dim) {
        *t2Offset += curDimIndex * t2.strides[d];
      }
      linearId /= index.sizes[d];
    }
  }
};

// Same as above but using a dynamic number of dimensions.
template <typename IndexType, typename Ntype>
struct IndexToScatterGatherOffsets<IndexType, Ntype, -1> {
  static __device__ void compute(
      IndexType linearId, const int dim,
      const TensorInfo<int64_t, IndexType>& index, IndexType* indexOffset,
      const TensorInfo<Ntype, IndexType>& t1, IndexType* t1Offset,
      const TensorInfo<Ntype, IndexType>& t2, IndexType* t2Offset) {
    for (int d = index.dims - 1; d >= 0; d--) {
      IndexType curDimIndex = linearId % index.sizes[d];
      *indexOffset += curDimIndex * index.strides[d];
      *t1Offset += curDimIndex * t1.strides[d];
      if (d != dim) {
        *t2Offset += curDimIndex * t2.strides[d];
      }
      linearId /= index.sizes[d];
    }
  }

  static __device__ void compute(
      IndexType linearId, const int dim,
      const TensorInfo<int64_t, IndexType>& index, IndexType* indexOffset,
      const TensorInfo<Ntype, IndexType>& t2, IndexType* t2Offset) {
    for (int d = index.dims - 1; d >= 0; d--) {
      IndexType curDimIndex = linearId % index.sizes[d];
      *indexOffset += curDimIndex * index.strides[d];
      if (d != dim) {
        *t2Offset += curDimIndex * t2.strides[d];
      }
      linearId /= index.sizes[d];
    }
  }
};

template <typename IndexType, typename Ntype, int Dims>
__global__ void THCudaTensor_gatherKernel(
    TensorInfo<Ntype, IndexType> tensor,
    TensorInfo<Ntype, IndexType> src,
    TensorInfo<int64_t, IndexType> index,
    const int dim,
    const IndexType totalElements) {
  for (IndexType linearId = blockIdx.x * blockDim.x + threadIdx.x;
       linearId < totalElements;
       linearId += gridDim.x * blockDim.x) {
    IndexType tensorOffset = 0;
    IndexType srcOffset = 0;
    IndexType indexOffset = 0;

    IndexToScatterGatherOffsets<IndexType, Ntype, Dims>::compute(linearId, dim,
                                                          index, &indexOffset,
                                                          tensor, &tensorOffset,
                                                          src, &srcOffset);

    int64_t indexValue = index.data[indexOffset] - TH_INDEX_BASE;
    assert(indexValue >= 0 && indexValue < src.sizes[dim]);
    srcOffset += indexValue * src.strides[dim];

    tensor.data[tensorOffset] = src.data[srcOffset];
  }
}

template <typename IndexType, typename Ntype, int Dims>
__global__ void THCudaTensor_scatterKernel(
    TensorInfo<Ntype, IndexType> tensor,
    TensorInfo<Ntype, IndexType> src,
    TensorInfo<int64_t, IndexType> index,
    const int dim,
    const IndexType totalElements) {
  for (IndexType linearId = blockIdx.x * blockDim.x + threadIdx.x;
       linearId < totalElements;
       linearId += gridDim.x * blockDim.x) {
    IndexType tensorOffset = 0;
    IndexType srcOffset = 0;
    IndexType indexOffset = 0;

    IndexToScatterGatherOffsets<IndexType, Ntype, Dims>::compute(linearId, dim,
                                                          index, &indexOffset,
                                                          src, &srcOffset,
                                                          tensor, &tensorOffset);

    int64_t indexValue = index.data[indexOffset] - TH_INDEX_BASE;
    assert(indexValue >= 0 && indexValue < tensor.sizes[dim]);
    tensorOffset += indexValue * tensor.strides[dim];

    tensor.data[tensorOffset] = src.data[srcOffset];
  }
}

template <typename IndexType, typename Ntype, int Dims>
__global__ void THCudaTensor_scatterAddKernel(
    TensorInfo<Ntype, IndexType> tensor,
    TensorInfo<Ntype, IndexType> src,
    TensorInfo<int64_t, IndexType> index,
    const int dim,
    const IndexType totalElements) {
  for (IndexType linearId = blockIdx.x * blockDim.x + threadIdx.x;
       linearId < totalElements;
       linearId += gridDim.x * blockDim.x) {
    IndexType tensorOffset = 0;
    IndexType srcOffset = 0;
    IndexType indexOffset = 0;

    IndexToScatterGatherOffsets<IndexType, Ntype, Dims>::compute(linearId, dim,
                                                          index, &indexOffset,
                                                          src, &srcOffset,
                                                          tensor, &tensorOffset);

    int64_t indexValue = index.data[indexOffset] - TH_INDEX_BASE;
    assert(indexValue >= 0 && indexValue < tensor.sizes[dim]);
    tensorOffset += indexValue * tensor.strides[dim];

    atomicAdd(&tensor.data[tensorOffset], src.data[srcOffset]);
  }
}

template <typename IndexType, typename Ntype, int Dims>
__global__ void THCudaTensor_scatterFillKernel(
    TensorInfo<Ntype, IndexType> tensor,
    TensorInfo<int64_t, IndexType> index,
    Ntype value,
    const int dim,
    const IndexType totalElements) {
  for (IndexType linearId = blockIdx.x * blockDim.x + threadIdx.x;
       linearId < totalElements;
       linearId += gridDim.x * blockDim.x) {
    IndexType tensorOffset = 0;
    IndexType indexOffset = 0;

    IndexToScatterGatherOffsets<IndexType, Ntype, Dims>::compute(linearId, dim,
                                                          index, &indexOffset,
                                                          tensor, &tensorOffset);

    int64_t indexValue = index.data[indexOffset] - TH_INDEX_BASE;
    assert(indexValue >= 0 && indexValue < tensor.sizes[dim]);
    tensorOffset += indexValue * tensor.strides[dim];

    tensor.data[tensorOffset] = value;
  }
}

#include "generic/THCTensorScatterGather.cu"
#include "THCGenerateAllTypes.h"
