
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/L1Cost.cu"
#else

void THNN_(L1Cost_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output)
{
  THCUNN_check_dim_size(state, output, 1, 0, 1);
  THCUNN_assertSameGPU(state, 1, input);
  accntype sum;
  ptrdiff_t size = THCTensor_(nElement)(state, input);
  input = THCTensor_(newContiguous)(state, input);
  thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
  sum = thrust::transform_reduce(input_data, input_data+size, l1cost_functor<ntype, accntype>(), accntype(0), thrust::plus<accntype>());

  THCTensor_(free)(state, input);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accntype, ntype>::to(sum));
}

void THNN_(L1Cost_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput)
{
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 2, input, gradInput);
  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  THCTensor_(resizeAs)(state, gradInput, input);

  thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<ntype> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, gradInput_data, l1cost_updateGradInput_functor<ntype>());

  THCTensor_(free)(state, input);
}

#endif
