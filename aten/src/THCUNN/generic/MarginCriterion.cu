
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MarginCriterion.cu"
#else

void THNN_(MarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage,
           accntype margin_)
{
  ntype margin = ScalarConvert<accntype, ntype>::to(margin_);
  THCUNN_check_nElement(state, input, target);
  THCUNN_check_dim_size(state, output, 1, 0, 1);
  THCUNN_assertSameGPU(state, 2, input, target);

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<ntype> target_data(THCTensor_(data)(state, target));
  accntype sum = thrust::inner_product(input_data, input_data+size, target_data, (accntype) 0, thrust::plus<accntype>(),
      margin_functor<ntype, accntype>(ScalarConvert<ntype, accntype>::to(margin)));

  if (sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accntype, ntype>::to(sum));
}


void THNN_(MarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradInput,
           bool sizeAverage,
           accntype margin_)
{
  ntype margin = ScalarConvert<accntype, ntype>::to(margin_);

  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, gradInput);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  accntype norm = sizeAverage ? 1.f/size : 1;

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  THCTensor_(resizeAs)(state, gradInput, input);

  thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<ntype> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<ntype> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data,
      margin_updateGradInput_functor<ntype, accntype>(ScalarConvert<ntype, accntype>::to(margin), norm));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif
