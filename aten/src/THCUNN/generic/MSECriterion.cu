
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MSECriterion.cu"
#else

#include "THCApply.cuh"

void THNN_(MSECriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage,
           bool reduce)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, output);

  if (reduce) {
    THCTensor_(resize1d)(state, output, 1);

    ptrdiff_t size = THCTensor_(nElement)(state, input);

    input = THCTensor_(newContiguous)(state, input);
    target = THCTensor_(newContiguous)(state, target);

    THCThrustAllocator thrustAlloc(state);
    thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
    thrust::device_ptr<ntype> target_data(THCTensor_(data)(state, target));
    accntype sum = thrust::inner_product(
#if CUDA_VERSION >= 7000
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      input_data, input_data+size, target_data, (accntype) 0,
      thrust::plus<accntype>(), mse_functor<ntype, accntype>());

    if (sizeAverage)
      sum /= size;

    THCTensor_(free)(state, input);
    THCTensor_(free)(state, target);

    THCTensor_(set1d)(state, output, 0, ScalarConvert<accntype, ntype>::to(sum));
    return;
  }

  THCTensor_(resizeAs)(state, output, input);
  THC_pointwiseApply3(
      state,
      input,
      target,
      output,
      mse_updateOutput_functor<ntype>());
}

void THNN_(MSECriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           bool sizeAverage,
           bool reduce)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 4, input, target, gradInput, gradOutput);

  if (reduce) {
    ptrdiff_t size = THCTensor_(nElement)(state, input);

    THCUNN_check_dim_size(state, gradOutput, 1, 0, 1);
    accntype norm = sizeAverage ? (accntype)(2)/size : (accntype)(2);
    norm *= ScalarConvert<ntype, accntype>::to(THCTensor_(get1d)(state, gradOutput, 0));

    input = THCTensor_(newContiguous)(state, input);
    target = THCTensor_(newContiguous)(state, target);

    THCTensor_(resizeAs)(state, gradInput, input);

    THCThrustAllocator thrustAlloc(state);
    thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
    thrust::device_ptr<ntype> target_data(THCTensor_(data)(state, target));
    thrust::device_ptr<ntype> gradInput_data(THCTensor_(data)(state, gradInput));

    thrust::transform(
#if CUDA_VERSION >= 7000
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      input_data, input_data+size, target_data, gradInput_data,
      mse_updateGradInput_functor<ntype, accntype>(norm));

    THCTensor_(free)(state, input);
    THCTensor_(free)(state, target);
    return;
  }

  THCUNN_check_shape(state, input, gradOutput);
  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  THCTensor_(resizeAs)(state, gradInput, input);

  THCThrustAllocator thrustAlloc(state);
  thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<ntype> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<ntype> gradOutput_data(THCTensor_(data)(state, gradOutput));
  thrust::device_ptr<ntype> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    input_data, input_data+size, target_data, gradInput_data,
    mse_updateGradInput_functor<ntype, accntype>(2));

  thrust::transform(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    gradInput_data, gradInput_data+size, gradOutput_data, gradInput_data,
    thrust::multiplies<ntype>());

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
  THCTensor_(free)(state, gradOutput);
}

#endif
