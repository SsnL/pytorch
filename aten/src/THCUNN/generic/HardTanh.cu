
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/HardTanh.cu"
#else

#include "../common.h"

void THNN_(HardTanh_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           accntype min_val_,
           accntype max_val_,
           bool inplace)
{
  ntype min_val = ScalarConvert<accntype, ntype>::to(min_val_);
  ntype max_val = ScalarConvert<accntype, ntype>::to(max_val_);

  THCUNN_assertSameGPU(state, 2, input, output);
  if(inplace)
  {
    THCTensor_(set)(state, output, input);
    THC_pointwiseApply1(state, output, hardtanhupdateOutput_functor<ntype>(min_val, max_val));
  }
  else
  {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply2(state, output, input,
                               hardtanhupdateOutput_functor<ntype>(min_val, max_val));
  }
}

void THNN_(HardTanh_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           accntype min_val_,
           accntype max_val_,
           bool inplace)
{
  ntype min_val = ScalarConvert<accntype, ntype>::to(min_val_);
  ntype max_val = ScalarConvert<accntype, ntype>::to(max_val_);

  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 3, input, gradOutput, gradInput);

  if (inplace)
  {
    THCTensor_(set)(state, gradInput, gradOutput);
    THC_pointwiseApply2(state, gradInput, input,
                                 hardtanhupdateGradInput_functor<ntype>(min_val, max_val));
  }
  else
  {
    THCTensor_(resizeAs)(state, gradInput, input);
    THC_pointwiseApply3(state, gradInput, input, gradOutput,
                                 hardtanhupdateGradInput_functor<ntype>(min_val, max_val));
  }
}

#endif
