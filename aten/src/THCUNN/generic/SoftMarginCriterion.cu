
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SoftMarginCriterion.cu"
#else

void THNN_(SoftMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage)
{
  THCUNN_check_nElement(state, input, target);
  THCTensor_(resize1d)(state, output, 1);
  THCUNN_assertSameGPU(state, 2, input, target);
  accntype sum;

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<ntype> target_data(THCTensor_(data)(state, target));
  sum = thrust::inner_product(input_data, input_data+size, target_data, (accntype) 0, thrust::plus<accntype>(), softmargin_functor<ntype, accntype>());

  if(sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accntype, ntype>::to(sum));
}

void THNN_(SoftMarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradInput,
           bool sizeAverage)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, gradInput);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  accntype norm = (sizeAverage ? 1./size : 1.);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  THCTensor_(resizeAs)(state, gradInput, input);

  thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<ntype> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<ntype> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data, softmargin_updateGradInput_functor<ntype, accntype>(norm));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif
