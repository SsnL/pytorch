
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SoftPlus.cu"
#else

#include "../common.h"

void THNN_(SoftPlus_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           accntype beta_,
           accntype threshold_)
{
  ntype beta = ScalarConvert<accntype, ntype>::to(beta_);
  ntype threshold = ScalarConvert<accntype, ntype>::to(threshold_);
  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(resizeAs)(state, output, input);
  THC_pointwiseApply2(state, output, input, softPlusupdateOutput_functor<ntype>(threshold, beta));
}

void THNN_(SoftPlus_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output,
           accntype beta_,
           accntype threshold_)
{
  ntype beta = ScalarConvert<accntype, ntype>::to(beta_);
  ntype threshold = ScalarConvert<accntype, ntype>::to(threshold_);
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 4, input, output, gradOutput, gradInput);
  THCTensor_(resizeAs)(state, gradInput, output);
  THC_pointwiseApply3(state, gradInput, output, gradOutput, softPlusupdateGradInput_functor<ntype>(threshold, beta));
}

#endif
