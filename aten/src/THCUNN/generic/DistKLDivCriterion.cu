
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/DistKLDivCriterion.cu"
#else

#include "THCApply.cuh"

void THNN_(DistKLDivCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage,
           bool reduce)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 2, input, target);

  THArgCheck(THCTensor_(nElement)(state, input) == THCTensor_(nElement)(state, target), 2,
             "input and target need to have the same number of elements");

  if (!reduce) {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply3(state, input, target, output,
                        kl_updateOutput_no_reduce_functor<ntype>());
    return;
  }

  THCTensor_(resize1d)(state, output, 1);

  accntype sum;

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<ntype> target_data(THCTensor_(data)(state, target));
  sum = thrust::inner_product(input_data, input_data+size, target_data, (accntype) 0, thrust::plus<accntype>(), kl_functor<ntype, accntype>());

  if (sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accntype, ntype>::to(sum));
}

void THNN_(DistKLDivCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           bool sizeAverage,
           bool reduce)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 4, input, target, gradInput, gradOutput);

  THArgCheck(THCTensor_(nElement)(state, input) == THCTensor_(nElement)(state, target), 2,
             "input and target need to have the same number of elements");

  THCTensor_(resizeAs)(state, gradInput, input);

  if (!reduce) {
    THCUNN_check_nElement(state, gradOutput, input);
    THC_pointwiseApply3(state, target, gradOutput, gradInput,
                        kl_updateGradInput_no_reduce_functor<ntype>());
    return;
  }

  THCUNN_check_dim_size(state, gradOutput, 1, 0, 1);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  ntype norm = (sizeAverage ? ScalarConvert<accntype, ntype>::to(accntype(1)/size) : ScalarConvert<int, ntype>::to(1));

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<ntype> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<ntype> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data,
                    kl_updateGradInput_functor<ntype>(norm, THCTensor_(get1d)(state, gradOutput, 0)));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif
