
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/ELU.cu"
#else

#include "../common.h"


void THNN_(ELU_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           accntype alpha,
           accntype scale,
           bool inplace)
{
  ntype negcoef = ScalarConvert<accntype, ntype>::to(alpha * scale);
  ntype poscoef = ScalarConvert<accntype, ntype>::to(scale);
  THCUNN_assertSameGPU(state, 2, input, output);

  if (inplace)
  {
    THC_pointwiseApply1(state, input, ELUupdateOutputIP_functor<ntype>(negcoef, poscoef));
    THCTensor_(set)(state, output, input);
  }
  else
  {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply2(state, output, input, ELUupdateOutput_functor<ntype>(negcoef, poscoef));
  }
}


void THNN_(ELU_updateGradInput)(
           THCState *state,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output,
           accntype alpha,
           accntype scale)
{
  ntype negcoef = ScalarConvert<accntype, ntype>::to(alpha * scale);
  ntype poscoef = ScalarConvert<accntype, ntype>::to(scale);
  THCUNN_check_nElement(state, output, gradOutput);
  THCUNN_assertSameGPU(state, 3, output, gradOutput, gradInput);

  THCTensor_(resizeAs)(state, gradInput, output);
  THC_pointwiseApply3(state, gradInput, output, gradOutput, ELUupdateGradInput_functor<ntype>(negcoef, poscoef));
}

#endif
