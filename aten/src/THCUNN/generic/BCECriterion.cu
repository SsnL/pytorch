
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/BCECriterion.cu"
#else

#include "THCApply.cuh"

void THNN_(BCECriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage,
           THCTensor *weights,
           bool reduce)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_check_nElement(state, input, weights);
  THCUNN_assertSameGPU(state, 3, input, target, weights);

  if (!reduce) {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply3(state, input, target, output,
        bce_updateOutput_no_reduce_functor<ntype, accntype>());
    if (weights) {
      THCTensor_(cmul)(state, output, output, weights);
    }
    return;
  }

  THCTensor_(resize1d)(state, output, 1);
  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<ntype> target_data(THCTensor_(data)(state, target));

  accntype sum;
  if (weights) {
    weights = THCTensor_(newContiguous)(state, weights);
    thrust::device_ptr<ntype> weights_data(THCTensor_(data)(state, weights));
    sum = thrust::transform_reduce(
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data, weights_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size, weights_data+size)),
      bce_functor_weights<ntype, accntype>(),
      (accntype) 0,
      thrust::plus<accntype>()
    );
    THCTensor_(free)(state, weights);
  } else {
    sum = thrust::transform_reduce(
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size)),
      bce_functor<ntype, accntype>(),
      (accntype) 0,
      thrust::plus<accntype>()
    );
  }

  if (sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accntype, ntype>::to(sum));
}

void THNN_(BCECriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           bool sizeAverage,
           THCTensor *weights,
           bool reduce)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_check_nElement(state, input, weights);
  THCUNN_assertSameGPU(state, 4, input, target, gradInput, weights);

  THCTensor_(resizeAs)(state, gradInput, input);

  if (!reduce) {
    THCUNN_check_nElement(state, gradOutput, input);
    THC_pointwiseApply3(state, input, target, gradInput,
        bce_updateGradInput_no_reduce_functor<ntype, accntype>());
    THCTensor_(cmul)(state, gradInput, gradInput, gradOutput);
    if (weights) {
      THCTensor_(cmul)(state, gradInput, gradInput, weights);
    }
    return;
  }

  THCUNN_check_dim_size(state, gradOutput, 1, 0, 1);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  ntype norm = ScalarConvert<accntype, ntype>::to((sizeAverage ? accntype(1)/size : accntype(1)) * THCTensor_(get1d)(state, gradOutput, 0));

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<ntype> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<ntype> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<ntype> gradInput_data(THCTensor_(data)(state, gradInput));

  if (weights) {
    weights = THCTensor_(newContiguous)(state, weights);
    thrust::device_ptr<ntype> weights_data(THCTensor_(data)(state, weights));
    thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data, weights_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size, weights_data+size)),
      gradInput_data,
      bce_updateGradInput_functor_weights<ntype, accntype>(norm)
    );
    THCTensor_(free)(state, weights);
  } else {
    thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size)),
      gradInput_data,
      bce_updateGradInput_functor<ntype, accntype>(norm)
    );
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif
