
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/LeakyReLU.cu"
#else

#include "../common.h"

void THNN_(LeakyReLU_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           accntype negval_,
           bool inplace)
{
  ntype negval = ScalarConvert<accntype, ntype>::to(negval_);

  THCUNN_assertSameGPU(state, 2, input, output);

  if (inplace)
  {
    THC_pointwiseApply1(state, input, LeakyReLUUpdateOutputIP<ntype>(negval));
    THCTensor_(set)(state, output, input);
  }
  else
  {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply2(state, output, input, LeakyReLUUpdateOutput<ntype>(negval));
  }

  THCudaCheck(cudaGetLastError());
}

void THNN_(LeakyReLU_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           accntype negval_,
           bool inplace)
{
  ntype negval = ScalarConvert<accntype, ntype>::to(negval_);

  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 3, input, gradInput, gradOutput);

  if (inplace)
  {
    THC_pointwiseApply2(state, gradOutput, input, LeakyReLUUpdateGradInputIP<ntype>(negval));
    THCTensor_(set)(state, gradInput, gradOutput);
  }
  else
  {
    THCTensor_(resizeAs)(state, gradInput, input);
    THC_pointwiseApply3(state, gradInput, input, gradOutput, LeakyReLUUpdateGradInput<ntype>(negval));
  }

  THCudaCheck(cudaGetLastError());
}

#endif
