
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/Threshold.cu"
#else

#include "../common.h"

void THNN_(Threshold_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           accntype threshold_,
           accntype val_,
           bool inplace)
{
  ntype threshold = ScalarConvert<accntype, ntype>::to(threshold_);
  ntype val = ScalarConvert<accntype, ntype>::to(val_);
  THCUNN_assertSameGPU(state, 2, input, output);

  if (inplace)
  {
    THC_pointwiseApply1(state, input,
      ThresholdUpdateOutputIP<ntype>(threshold, val)
    );
    THCTensor_(set)(state, output, input);
  }
  else
  {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply2(state, output, input,
      ThresholdUpdateOutput<ntype>(threshold, val)
    );
  }

  THCudaCheck(cudaGetLastError());
}

void THNN_(Threshold_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           accntype threshold_,
           accntype val_,
           bool inplace)
{
  ntype threshold = ScalarConvert<accntype, ntype>::to(threshold_);
  ntype val = ScalarConvert<accntype, ntype>::to(val_);
  (void) val;
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 3, input, gradInput, gradOutput);

  if (inplace)
  {
    THC_pointwiseApply2(state, gradOutput, input,
      ThresholdUpdateGradInputIP<ntype>(threshold)
    );
    THCTensor_(set)(state, gradInput, gradOutput);
  }
  else
  {
    THCTensor_(resizeAs)(state, gradInput, input);
    THC_pointwiseApply3(state, gradInput, input, gradOutput,
       ThresholdUpdateGradInput<ntype>(threshold)
    );
  }

  THCudaCheck(cudaGetLastError());
}

#endif
