
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialUpSamplingBilinear.cu"
#else

static inline void THNN_(SpatialUpSamplingBilinear_shapeCheck)
                        (THCState *state,
                         THCTensor *input, THCTensor *gradOutput,
                         int nBatch, int nChannels,
                         int inputHeight, int inputWidth,
                         int outputHeight, int outputWidth) {
  THArgCheck(inputHeight > 0 && inputWidth > 0
             && outputHeight > 0 && outputWidth > 0, 2,
             "input and output sizes should be greater than 0,"
             " but got input (H: %d, W: %d) output (H: %d, W: %d)",
             inputHeight, inputWidth, outputHeight, outputWidth);
  if (input != NULL) {
     THCUNN_argCheck(state, input->nDimension == 4, 2, input,
                     "4D input tensor expected but got: %s");
  }

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, 4, 0, nBatch);
    THCUNN_check_dim_size(state, gradOutput, 4, 1, nChannels);
    THCUNN_check_dim_size(state, gradOutput, 4, 2, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, 4, 3, outputWidth);
  }
}

void THNN_(SpatialUpSamplingBilinear_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int outputHeight,
           int outputWidth)
{
  int nbatch = THCTensor_(size)(state, input, 0);
  int channels = THCTensor_(size)(state, input, 1);
  int inputHeight = THCTensor_(size)(state, input, 2);
  int inputWidth = THCTensor_(size)(state, input, 3);
  THNN_(SpatialUpSamplingBilinear_shapeCheck)
       (state, input, NULL,
        nbatch, channels,
        inputHeight, inputWidth,
        outputHeight, outputWidth);
  input = THCTensor_(newContiguous)(state, input);
  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(resize4d)(state, output,
                       THCTensor_(size)(state, input, 0),
                       THCTensor_(size)(state, input, 1),
                       outputHeight, outputWidth);
  THCTensor_(zero)(state, output);
  THCDeviceTensor<ntype, 4> idata = toDeviceTensor<ntype, 4>(state, input);
  THCDeviceTensor<ntype, 4> odata = toDeviceTensor<ntype, 4>(state, output);
  THAssert(inputHeight > 0 && inputWidth > 0 && outputHeight > 0 && outputWidth > 0);
  const accntype rheight= (outputHeight > 1) ? (accntype)(inputHeight - 1)/(outputHeight - 1) : accntype(0);
  const accntype rwidth = (outputWidth > 1) ? (accntype)(inputWidth - 1)/(outputWidth - 1) : accntype(0);
  const int num_kernels = outputHeight * outputWidth;
  const int num_threads =
    THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock;
  cudaStream_t stream = THCState_getCurrentStream(state);
  caffe_gpu_interp2_kernel<ntype, accntype> <<<THCCeilDiv(num_kernels, num_threads), num_threads ,
   0 , stream>>>(num_kernels, rheight, rwidth, idata, odata);
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, input);
}


void THNN_(SpatialUpSamplingBilinear_updateGradInput)(
           THCState *state,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int nbatch,
           int nchannels,
           int inputHeight,
           int inputWidth,
           int outputHeight,
           int outputWidth)
{
  THNN_(SpatialUpSamplingBilinear_shapeCheck)
       (state, NULL, gradOutput,
        nbatch, nchannels,
        inputHeight, inputWidth,
        outputHeight, outputWidth);
  gradInput = THCTensor_(newContiguous)(state, gradInput);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCUNN_assertSameGPU(state, 2, gradOutput, gradInput);
  THCTensor_(resize4d)(state, gradInput, nbatch, nchannels, inputHeight, inputWidth);
  THCTensor_(zero)(state, gradInput);
  THCDeviceTensor<ntype, 4> data1 = toDeviceTensor<ntype, 4>(state, gradInput);
  THCDeviceTensor<ntype, 4> data2 = toDeviceTensor<ntype, 4>(state, gradOutput);
  int height1 = data1.getSize(2);
  int width1 = data1.getSize(3);
  int height2 = data2.getSize(2);
  int width2 = data2.getSize(3);
  assert(height1 > 0 && width1 > 0 && height2 > 0 && width2 > 0);
  const accntype rheight= (height2 > 1) ? (accntype)(height1 - 1)/(height2 - 1) : accntype(0);
  const accntype rwidth = (width2 > 1) ? (accntype)(width1 - 1) / (width2 - 1) : accntype(0);
  const int num_kernels = height2 * width2;
  const int num_threads =
    THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock;
  cudaStream_t stream = THCState_getCurrentStream(state);
  caffe_gpu_interp2_kernel_backward<ntype ,accntype> <<<THCCeilDiv(num_kernels, num_threads),
  num_threads, 0, stream>>>(num_kernels, rheight, rwidth, data1, data2);
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, gradInput);
  THCTensor_(free)(state, gradOutput);
}

#endif
